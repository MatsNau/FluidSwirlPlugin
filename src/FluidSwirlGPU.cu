#include <hip/hip_runtime.h>

#include <math.h>

// CUDA kernel for GPU-accelerated swirl processing
__global__ void fluidSwirlKernel(
    unsigned char* dst, 
    const unsigned char* src,
    int width, 
    int height,
    float centerX, 
    float centerY,
    float swirlIntensity,
    float decay,
    int nComponents)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    // Calculate distance from swirl center
    float dx = x - centerX;
    float dy = y - centerY;
    float distance = sqrtf(dx * dx + dy * dy);
    
    // Apply swirl transformation
    float angle = atan2f(dy, dx);
    float swirlAngle = swirlIntensity * expf(-distance / decay);
    angle += swirlAngle;
    
    // Calculate source coordinates with subpixel precision
    float srcX = centerX + distance * cosf(angle);
    float srcY = centerY + distance * sinf(angle);
    
    int dstIdx = (y * width + x) * nComponents;
    
    // Bilinear interpolation for smooth results
    if (srcX >= 0 && srcX < width-1 && srcY >= 0 && srcY < height-1) {
        int x0 = (int)floorf(srcX);
        int y0 = (int)floorf(srcY);
        int x1 = x0 + 1;
        int y1 = y0 + 1;
        
        float fx = srcX - x0;
        float fy = srcY - y0;
        float fx1 = 1.0f - fx;
        float fy1 = 1.0f - fy;
        
        // Interpolation weights
        float w00 = fx1 * fy1;
        float w10 = fx * fy1;
        float w01 = fx1 * fy;
        float w11 = fx * fy;
        
        for (int c = 0; c < nComponents; c++) {
            int src00 = (y0 * width + x0) * nComponents + c;
            int src10 = (y0 * width + x1) * nComponents + c;
            int src01 = (y1 * width + x0) * nComponents + c;
            int src11 = (y1 * width + x1) * nComponents + c;
            
            float result = w00 * src[src00] + 
                          w10 * src[src10] + 
                          w01 * src[src01] + 
                          w11 * src[src11];
            
            dst[dstIdx + c] = (unsigned char)fminf(255.0f, fmaxf(0.0f, result));
        }
    } else {
        // Black for out-of-bounds pixels
        for (int c = 0; c < nComponents; c++) {
            dst[dstIdx + c] = 0;
        }
    }
}

// Host function to launch CUDA kernel
extern "C" void launchSwirlKernel(
    unsigned char* d_dst,
    const unsigned char* d_src,
    int width,
    int height,
    float centerX,
    float centerY,
    float swirlIntensity,
    float decay,
    int nComponents)
{
    // Configure CUDA grid and block dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);
    
    // Launch kernel
    fluidSwirlKernel<<<gridSize, blockSize>>>(
        d_dst, d_src, width, height,
        centerX, centerY, swirlIntensity, decay, nComponents);
    
    // Wait for kernel to complete
    hipDeviceSynchronize();
}

// Memory management helpers
extern "C" void* allocateGPUMemory(size_t size)
{
    void* ptr;
    hipMalloc(&ptr, size);
    return ptr;
}

extern "C" void freeGPUMemory(void* ptr)
{
    hipFree(ptr);
}

extern "C" void copyToGPU(void* dst, const void* src, size_t size)
{
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

extern "C" void copyFromGPU(void* dst, const void* src, size_t size)
{
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

// Check if CUDA is available
extern "C" bool isCudaAvailable()
{
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    return (error == hipSuccess && deviceCount > 0);
}